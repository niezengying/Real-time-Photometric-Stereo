#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//////////////////////   Calculate Z functions  :  CUDA C   /////////////////////////
////////////////////////////////////////////////////////////////////////////

#include "newPro.h"

#include <helper_functions.h> 
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include ""
#include <timer.h>

// Reconstruction data
extern uchar1 *dev_Mask, *dev_Src;  
extern double *dev_wx, *dev_wy,*dev_Z;
extern hipDoubleComplex *dev_Zx, *dev_Zy, *dev_Cw ;
extern	float4* dev_gl_Z ;

extern 	hipfftHandle pland2zP, pland2zQ, planz2d_inv;
extern hipblasHandle_t handle_Z_min,handle_Z_max;
extern int minZ_idx,maxZ_idx;

// BMP data
extern int imageW, imageH, bmpHdrOff;
extern unsigned char *h_Src,*h_Mask;
extern char *imagePath[];
extern char *maskPath;

// Run time
hipEvent_t start, stop;
float elapsedTime;

////�豸�ڴ��������ͷ�
/////////////////////Malloc && Free//////////////////////////////
extern "C" void CudaMalloc()
{
	checkCudaErrors(hipSetDevice(0));
	printf("CUDA Malloc...\n");
	
	//malloc Mask
	checkCudaErrors(hipMalloc((void**)&dev_Mask,imageW*imageH*sizeof(uchar1)));
	checkCudaErrors(hipMemset(dev_Mask,0,imageH*imageW*sizeof(uchar1)));

	//malloc Src
	checkCudaErrors(hipMalloc((void**)&dev_Src,imageW*imageH*sizeof(uchar1)*IMGNUM));
	checkCudaErrors(hipMemset(dev_Src,0,imageH*imageW*sizeof(uchar1)*IMGNUM));

	//malloc wx && wy
	checkCudaErrors(hipMalloc((void**)&dev_wx, imageH*imageW*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&dev_wy, imageH*imageW*sizeof(double)));
	//hipChannelFormatDesc desc_wxy = hipCreateChannelDesc<double>();
	//hipBindTexture(NULL,tex_wx,dev_wx,desc_wxy,imageH*imageW*sizeof(double));
	//hipBindTexture(NULL,tex_wy,dev_wy,desc_wxy,imageH*imageW*sizeof(double));
	checkCudaErrors(hipMemset(dev_wx,0,imageH*imageW*sizeof(double)));
	checkCudaErrors(hipMemset(dev_wy,0,imageH*imageW*sizeof(double)));

	//malloc dev_fourier_pq 
	hipMalloc((void**)&dev_Zx,sizeof(hipDoubleComplex)*imageW*(imageH));
	hipMalloc((void**)&dev_Zy,sizeof(hipDoubleComplex)*imageW*(imageH));
	checkCudaErrors(hipMemset(dev_Zx,0,sizeof(hipDoubleComplex)*imageW*(imageH)));
	checkCudaErrors(hipMemset(dev_Zy,0,sizeof(hipDoubleComplex)*imageW*(imageH)));

	//malloc dev_Cw
	checkCudaErrors(hipMalloc((void**)&dev_Cw, (imageH)*imageW*sizeof(	hipDoubleComplex)));
	checkCudaErrors(hipMemset(dev_Cw,0, (imageH)*imageW*sizeof(hipDoubleComplex)));
	
	//malloc dev_Z
	checkCudaErrors(hipMalloc((void**)&dev_Z, imageH*imageW*sizeof(double)));
	checkCudaErrors(hipMemset(dev_Z,0, imageH*imageW*sizeof(double)));

	hipEventCreate(&start);
	hipEventCreate(&stop);

	LoadBMPFile(&h_Mask, &bmpHdrOff, &imageW, &imageH, maskPath);
	LoadManyBmp(&h_Src,bmpHdrOff,imageW, imageH,imagePath);
	printf("%d, %d\n",imageW,imageH);

}

extern "C" void CudaFree()
{
	hipFree(dev_Mask);
	hipFree(dev_Src);
	hipFree(dev_wx);
	hipFree(dev_wy);
	hipFree(dev_Zx);
	hipFree(dev_Zy);
	hipFree(dev_Cw);
	hipFree(dev_Z);
}

extern "C" void load_Image()
{
	LoadBMPFile(&h_Mask, &bmpHdrOff, &imageW, &imageH, maskPath);
	LoadManyBmp(&h_Src,bmpHdrOff,imageW, imageH,imagePath);
	checkCudaErrors(hipMemcpy(dev_Src,h_Src,imageW*imageH*sizeof(uchar1)*IMGNUM,hipMemcpyHostToDevice));
	free(h_Mask);
	free(h_Src);
}

///////////////////////Calculate the median Value//////////////////////
//count Wx && Wy
extern "C" void launch_count_Wxy(double *dev_wx,double *dev_wy, int width, int height)
{
	dim3 block_wxy(8,8,1);   //ÿ������߳���
	dim3 grid_wxy((width+block_wxy.x-1)/block_wxy.x, (height+block_wxy.y-1)/block_wxy.y,1); //ÿ����Ŀ���
	kernel_count_Wxy<<<grid_wxy,  block_wxy>>>(dev_wx,dev_wy, width, height);
}

//count Zx && Zy
extern "C" void launch_count_PQ(hipDoubleComplex *dev_Zx,hipDoubleComplex *dev_Zy,int width, int height)
{
	dim3	dimBlock(8,8,1);
    dim3	dimGrid((width+dimBlock.x-1)/dimBlock.x, (height+dimBlock.y-1)/dimBlock.y,1);
	kernel_count_pq<<<dimGrid,dimBlock>>>(dev_Zx,dev_Zy,dev_Mask,dev_Src,width,height);
}

//count C(w)
extern "C" void launch_count_Cw(hipDoubleComplex *dev_Cw, int width, int height)
{
	dim3 block_Cw(8,8,1);
	dim3 grid_Cw((width+block_Cw.x-1)/block_Cw.x, (height+block_Cw.y-1)/block_Cw.y,1);
	kernel_count_Cw<<<grid_Cw,  block_Cw>>>(dev_wx,dev_wy,dev_Zx,dev_Zy,dev_Cw, width, height);
}

//count the result Z
extern "C" void launch_count_z2gl(float4 *dev_gl_Z,int width, int height,float time)
{	
	dim3 block_z2gl(8,8,1);
	dim3 grid_z2gl((width+block_z2gl.x-1)/block_z2gl.x, (height+block_z2gl.y-1)/block_z2gl.y,1);
	kernel_copy_z2gl<<<grid_z2gl, block_z2gl>>>(dev_gl_Z,dev_Z, width, height,time);
}

////���Դ��벻�ù�ע
///////////////////////////////Test Code//////////////////////////////
extern "C" void test_cplx_SumMinMax(hipDoubleComplex *src,int width, int height)
{	
	double2 p,q;
	double sumP;
	checkCudaErrors(hipblasDzasum(handle_Z_min, width*height, src, 1, &sumP));
	checkCudaErrors(hipblasIzamin(handle_Z_min, width*height, src, 1, &minZ_idx));
	checkCudaErrors(hipblasIzamax(handle_Z_max,width*height, src, 1, &maxZ_idx));

	checkCudaErrors(hipMemcpy(&p,&src[minZ_idx-1],sizeof(double2),hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&q,&src[maxZ_idx-1],sizeof(double2),hipMemcpyDeviceToHost));

	//printf("Result:\t%f\t%f\t%f\n",sumP,abs(p.x)+abs(p.y),abs(q.x)+abs(q.y));
	printf("Result:\t%f\t%f\t%f\n",sumP,p.x,q.x);
}

extern "C" void test_dbl_SumMinMax(double *src,int width, int height)
{
	double p,q;
	double sumP;
	checkCudaErrors(hipblasDasum(handle_Z_min, width*height, src, 1, &sumP));
	checkCudaErrors(hipblasIdamin(handle_Z_min, width*height, src, 1, &minZ_idx));
	checkCudaErrors(hipblasIdamax(handle_Z_max,width*height, src, 1, &maxZ_idx));

	checkCudaErrors(hipMemcpy(&p,&src[minZ_idx-1],sizeof(double),hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&q,&src[maxZ_idx-1],sizeof(double),hipMemcpyDeviceToHost));

	printf("Result:\t%f\t%f\t%f\n",sumP,p,q);
}

template <class T, class T2>
 void test_rand_Print(T *src,T2 *src2,int width, int height)
{
	double *h_p, *h_p2;
	h_p = (double *)malloc(width*height*sizeof(T));
	h_p2 = (double *)malloc(width*height*sizeof(T2));
	hipMemcpy(h_p,src,width*height*sizeof(T),hipMemcpyDeviceToHost);
	hipMemcpy(h_p2,src2,width*height*sizeof(T2),hipMemcpyDeviceToHost);

	int ii = 0,jj = 0;
	int N1 = sizeof(T)/sizeof(double);
	int N2 = sizeof(T2)/sizeof(double);
	while((scanf("%d %d",&ii,&jj)) == 2 && ii != EOF)
	{
			ii = ii - 1 ;
			jj = jj - 1  ;
			if(N1>1)
				printf("%f\t%f\n",h_p[IDX2C(ii,jj,width) * N1], h_p[IDX2C(ii,jj,width)  * N1 + 1]);
			else
				printf("%f\t%f\n",h_p[IDX2C(ii,jj,width)]);

			if(N2>1)
				printf("%f\t%f\n", h_p2[IDX2C(ii,jj,width) * N2], h_p2[IDX2C(ii,jj,width)  * N2 + 1]);
			else
				printf("%f\t%f\n", h_p2[IDX2C(ii,jj,width)]);
	}

	free(h_p);
	free(h_p2);
}

 extern "C" void test_Print2File(double * src, int width, int height)
{
	double *h_p;		
	FILE *stream;
	h_p = (double *)malloc(width*height*sizeof(double));
	checkCudaErrors(hipMemcpy(h_p,src,width*height*sizeof(double),hipMemcpyDeviceToHost));
	if((stream = fopen("text.txt", "w+"))== NULL)
		printf("error!\n");

	printf("%d,%d\n",width,height);
	for(int j = 0; j < height ;j++)
		for(int i = 0; i < width; i++)
		{
			fprintf(stream,"%f\t%f\t%f\n",(float)i,(float)j,(float)(h_p[i+width*j])*(-0.5));
		}
	fclose(stream); 
	free(h_p);
}

 ///�����������
/////////////////////////Calculate Function//////////////////////
extern "C" void testCuda(struct hipGraphicsResource **cuda_vbo_resource,int width, int height,float g_fAnim)
{
	hipEventRecord(start,0);
	size_t size;

	//load_Image();
	checkCudaErrors(hipMemcpy(dev_Mask,h_Mask,width*height*sizeof(uchar1),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Src,h_Src,width*height*sizeof(uchar1)*IMGNUM,hipMemcpyHostToDevice));

	printf("\n0.	count  Wx && Wy !!! \n");
	printf("\n1.	count Zx && Zy !!! \n");
	launch_count_PQ(dev_Zx,dev_Zy,width, height);
	
	printf("\n2.	dev_pq FFT == Cx !!! \n");
	checkCudaErrors(hipfftExecZ2Z(pland2zP, dev_Zx, dev_Zx,HIPFFT_FORWARD));
	checkCudaErrors(hipfftExecZ2Z(pland2zQ, dev_Zy, dev_Zy,HIPFFT_FORWARD));

	printf("\n2.5.	Wxy  iFFTshift  !!! \n"); 
	printf("\n3.	count C(w) == Wx * Cx !!!\n");	//  Frankt-Chellappa Algrotihm
	launch_count_Cw(dev_Cw, width, height);	// Minimize Cw in this way
	//test_dbl_SumMinMax(dev_Cw,width, height);

	printf("\n4.	Z Reconstruction == C(w) IFFT !!!\n");
	checkCudaErrors(hipfftExecZ2Z(planz2d_inv, dev_Cw, dev_Cw,HIPFFT_BACKWARD));	// Reconstruction 
	hipMemcpy2D(dev_Z,sizeof(double),dev_Cw,sizeof(double2),sizeof(double),width*height,hipMemcpyDeviceToDevice);	//Get the real part of Z
	//test_rand_Print<double,double2>(dev_Z, dev_Cw, width, height);

	//checkCudaErrors(hipblasIdamax(handle_Z_max,width*height, dev_Z, 1, &maxZ_idx));

	printf("\n5.	bind	Z to opengl !!!\n");    ////����������OpenGL
	checkCudaErrors(hipGraphicsMapResources(1,cuda_vbo_resource,NULL));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&dev_gl_Z,&size,*cuda_vbo_resource));
	launch_count_z2gl(dev_gl_Z,width, height,g_fAnim);
	checkCudaErrors(hipGraphicsUnmapResources(1,cuda_vbo_resource,0)); 

	//Show the time
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("\ntime to generate:\t %3.1f ms\n\n",elapsedTime);
}